#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

// ------------[CUDA Kernel function to add elements of two arrays]---------------------
__global__ void add(int* a, int* b, int* c) 
{
    int index = threadIdx.x;
    c[index] = a[index] + b[index];
}

int main() 
{
    // Array size
    const int n = 10;
    int size = n * sizeof(int);

    // Host arrays [CPU]
    int h_a[n], h_b[n], h_c[n];

    // -------------------[Initialization]--------------------------
    for (int i = 0; i < n; i++)
{
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Device arrays [GPU]
    int* d_a, * d_b, * d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copying data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    /* 
    Define the number of threads per block and the number of blocks using dim3
    -----------------------------[ What is dim3? ]----------------------------------------
    The [dim3] data type in CUDA is used to define the dimensions of blocks and grids.
    It allows you to specify the number of threads in each block and the number of blocks in each grid.
    You can think of dim3 as a 3D vector with x, y, and z dimensions. In most simple cases 
    */

    dim3 threadsPerBlock(n, 1, 1);
    dim3 blocksPerGrid(1, 1, 1);

    //------------------------------[Launch kernel]-------------------------------
    add << <blocksPerGrid, threadsPerBlock >> > (d_a, d_b, d_c);

    // Copying the result back to host [CPU]
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // ----------------------[Display the results]---------------------------------
    for (int i = 0; i < n; i++) 
    {
        cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << endl;
    }

    // Free the device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
